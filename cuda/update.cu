
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>

#ifndef M_PI
#define M_PI 3.14159265359
#endif
#define MIN(a,b) (((a)<(b))?(a):(b))


typedef struct 
{
    double (*measurements)[2];
    int n_measurements;
    double *measurement_cov;
} landmark_measurements;

__device__ double mod_angle(double angle) {
    return atan2(sin(angle), cos(angle));
}

__device__ double vecnorm(double *v) {
    return sqrt(v[0]*v[0] + v[1]*v[1]);
}

__device__ bool in_sensor_range(double *position, double *landmark, double range, double fov) {
    double x = position[0];
    double y = position[1];
    double theta = position[2];
    double lx = landmark[0];
    double ly = landmark[1];

    double va[] = {lx - x, ly - y};
    double vb[] = {range * cos(theta), range * sin(theta)};

    if(vecnorm(va) > range) {
        return false;
    }

    double angle = acos(
        (va[0]*vb[0] + va[1]*vb[1])/(vecnorm(va)*vecnorm(vb))
    );

    if(angle <= (fov/2)) {
        return true;
    } else {
        return false;
    }
}

__device__ void to_coords(double *particle, double *in, double *out) {
    double x = particle[0];
    double y = particle[1];
    double theta = particle[2];

    double range = in[0];
    double bearing = in[1];

    out[0] = x + range * cos(bearing + theta);
    out[1] = y + range * sin(bearing + theta);
}

__device__ double* get_particle(double *particles, int i) {
    int max_landmarks = (int)particles[4];
    return (particles + (6 + 7*max_landmarks)*i);
}

__device__ double* get_mean(double *particle, int i)
{
    return (particle + 6 + 2*i);
}

__device__ double* get_cov(double *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 2*max_landmarks + 4*i);
}

__device__ double* get_landmark_prob(double *particle, int i)
{
    int max_landmarks = (int)particle[4];
    return (particle + 6 + 6*max_landmarks + i);
}

__device__ void increment_landmark_prob(double *particle, int i)
{
    int max_landmarks = (int)particle[4];
    double *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] += 1.0;
}

__device__ void decrement_landmark_prob(double *particle, int i)
{
    int max_landmarks = (int)particle[4];
    double *prob = (particle + 6 + 6*max_landmarks + i);
    prob[0] -= 1.0;
}

__device__ int get_n_landmarks(double *particle)
{
    return (int)particle[5];
}

__device__ void add_landmark(double *particle, double mean[2], double *cov)
{
    int n_landmarks = (int)particle[5];
    particle[5] = (double)(n_landmarks + 1);

    double *new_mean = get_mean(particle, n_landmarks);
    double *new_cov = get_cov(particle, n_landmarks);
    double *new_prob = get_landmark_prob(particle, n_landmarks);

    new_mean[0] = mean[0];
    new_mean[1] = mean[1];

    new_cov[0] = cov[0];
    new_cov[1] = cov[1];
    new_cov[2] = cov[2];
    new_cov[3] = cov[3];

    new_prob[0] = 1.0;
}

__device__ void remove_landmark(double *particle, int i)
{
    int n_landmarks = (int)particle[5];

    double *mean_a = get_mean(particle, i);
    double *mean_b = get_mean(particle, n_landmarks - 1);

    mean_a[0] = mean_b[0];
    mean_a[1] = mean_b[1];

    double *cov_a = get_cov(particle, i);
    double *cov_b = get_cov(particle, n_landmarks - 1);

    cov_a[0] = cov_b[0];
    cov_a[1] = cov_b[1];
    cov_a[2] = cov_b[2];
    cov_a[3] = cov_b[3];

    double *prob_a = get_landmark_prob(particle, i);
    double *prob_b = get_landmark_prob(particle, n_landmarks - 1);

    prob_a[0] = prob_b[0];
    
    particle[5] = (double)(n_landmarks - 1);
}

__device__ void vecmul(double *A, double *u, double *v)
{
    double a = A[0];
    double b = A[1];
    double c = A[2];
    double d = A[3];

    double e = u[0];
    double f = v[1];

    v[0] = a*e + b*f;
    v[1] = c*e + d*f;
}

__device__ void matmul(double *A, double *B, double *C)
{
    double a = A[0];
    double b = A[1];
    double c = A[2];
    double d = A[3];

    double e = B[0];
    double f = B[1];
    double g = B[2];
    double h = B[3];

    C[0] = a*e + b*g;
    C[1] = a*f + b*h;
    C[2] = c*e + d*g;
    C[3] = c*f + d*h;
}

__device__ void matmul_jacobian(double *H, double *E, double *R, double *S)
{
    double a = H[0];
    double b = H[1];
    double c = H[2];
    double d = H[3];

    double Ht[] = {
        a, c,
        b, d
    };

    matmul(H, E, S);
    matmul(S, Ht, S);

    S[0] += R[0];
    S[1] += R[1];
    S[2] += R[2];
    S[3] += R[3];
}

__device__ void pinv(double *A, double *B)
{
    double a = A[0];
    double b = A[1];
    double c = A[2];
    double d = A[3];

    double scalar = 1/(a*d - b*c);

    B[0] = scalar * d;
    B[1] = scalar * (-b);
    B[2] = scalar * (-c);
    B[3] = scalar * a;
}

__device__ double pdf(double *x, double *mean, double* cov)
{
    double cov_inv[] = {0, 0, 0, 0};
    pinv(cov, cov_inv);

    double scalar = 1/(2*M_PI*sqrt(cov[0]*cov[3] - cov[1]*cov[2]));

    double m = x[0] - mean[0];
    double n = mod_angle(x[1] - mean[1]);

    double arg = m*m*(cov_inv[0]) + n*n*(cov_inv[3]) + m*n*(cov_inv[1] + cov_inv[2]);

    return scalar * exp(-0.5 * arg);
}

__device__ void add_measurement_as_landmark(double *particle, double *measurement, double *measurement_cov)
{
    double pos[] = { particle[0], particle[1] };
    double landmark[] = {0, 0};
    to_coords(particle, measurement, landmark);

    double q = (landmark[0] - pos[0])*(landmark[0] - pos[0]) + (landmark[1] - pos[1])*(landmark[1] - pos[1]);

    double H[] = {
        (landmark[0] - pos[0])/(sqrt(q)), (landmark[1] - pos[1])/(sqrt(q)),
        -(landmark[1] - pos[1])/q, (landmark[0] - pos[0])/q
    };

    pinv(H, H);

    double H_inv_t[] = {
        H[0], H[2],
        H[1], H[3]
    };

    double S[] = {
        0, 0, 0, 0
    };

    matmul(H, measurement_cov, S);
    matmul(S, H_inv_t, S);
    add_landmark(particle, landmark, S);
}


__device__ void add_measurements_as_landmarks(double *particle, landmark_measurements *measurements)
{
    int n_measurements = measurements->n_measurements;
    double *measurement_cov = measurements->measurement_cov;

    for(int i = 0; i < n_measurements; i++) {
        add_measurement_as_landmark(particle, measurements->measurements[i], measurement_cov);
    }
}


__device__ double compute_dist(double *particle, int i, double *measurement, double *measurement_cov)
{
    double *landmark = get_mean(particle, i);
    double measurement_xy[] = {0, 0};
    to_coords(particle, measurement, measurement_xy);

    double dist = sqrt(
        (landmark[0] - measurement_xy[0])*(landmark[0] - measurement_xy[0]) +
        (landmark[1] - measurement_xy[1])*(landmark[1] - measurement_xy[1])
    );
    
    return dist;
}


__device__ void update_landmarks(int id, double *particle, landmark_measurements *measurements, int *in_range, int *n_matches, double range, double fov, double thresh)
{
    double *measurement_cov = measurements->measurement_cov;
    int n_measurements = measurements->n_measurements;

    int n_landmarks = get_n_landmarks(particle);

    int n_in_range = 0;
    for(int i = 0; i < n_landmarks; i++) {
        n_matches[i] = 0;
        double *mean = get_mean(particle, i);
        // in_range[n_in_range] = i;
        // n_in_range++;
        if(in_sensor_range(particle, mean, range, fov)) {
            in_range[n_in_range] = i;
            n_in_range++;
        }
    }

    for(int i = 0; i < n_measurements; i++) {
        double best = 1000000.0;
        int best_idx = -1;

        for(int j = 0; j < n_landmarks; j++) {
            double dist = compute_dist(particle, j, measurements->measurements[i], measurement_cov);

            if(dist <= thresh && dist < best /*&& n_matches[j] == 0*/) {
                best = dist;
                best_idx = j;
            }
        }

        if(best_idx != -1) {
            n_matches[best_idx]++;
        }

        if(best_idx != -1) {
            double *landmark = get_mean(particle, best_idx);
            double pos[] = { particle[0], particle[1] };
            double theta = particle[2];

            double q = (landmark[0] - pos[0])*(landmark[0] - pos[0]) + (landmark[1] - pos[1])*(landmark[1] - pos[1]);
            double measurement_predicted[] = {
                sqrt(q), mod_angle(atan2(landmark[1] - pos[1], landmark[0] - pos[0]) - theta)
            };

            double residual[2] = {
                measurements->measurements[i][0] - measurement_predicted[0],
                mod_angle(measurements->measurements[i][1] - measurement_predicted[1])
            };


            double H[] = {
                (landmark[0] - pos[0])/(sqrt(q)), (landmark[1] - pos[1])/(sqrt(q)),
                -(landmark[1] - pos[1])/q, (landmark[0] - pos[0])/q
            };

            double Ht[] = {
                H[0], H[2],
                H[1], H[3]
            };

            double S[] = {
                0, 0, 0, 0
            };

            double *landmark_cov = get_cov(particle, best_idx);
        
            matmul_jacobian(H, landmark_cov, measurement_cov, S);
            double S_inv[] = {0, 0, 0, 0};
            pinv(S, S_inv);


            double Q[] = {0, 0, 0, 0};
            double K[] = { 0, 0, 0, 0 };
            matmul(landmark_cov, Ht, Q);
            matmul(Q, S_inv, K);

            double K_residual[] = { 0, 0 };
            vecmul(K, residual, K_residual);
            landmark[0] += K_residual[0];
            landmark[1] += K_residual[1];

            double KH[] = { 0, 0, 0, 0};
            matmul(K, H, KH);
            double new_cov[] = { 1 - KH[0], -KH[1], -KH[2], 1 - KH[3] };
            matmul(new_cov, landmark_cov, new_cov);
            landmark_cov[0] = new_cov[0];
            landmark_cov[1] = new_cov[1];
            landmark_cov[2] = new_cov[2];
            landmark_cov[3] = new_cov[3];

            particle[3] *= pdf(measurements->measurements[i], measurement_predicted, S);

            increment_landmark_prob(particle, best_idx);

        } else {
            add_measurement_as_landmark(particle, measurements->measurements[i], measurement_cov);
        }
    }

    for(int i = n_in_range - 1; i > 0; i--) {
        int idx = in_range[i];
        if(n_matches[idx] == 0) {
            decrement_landmark_prob(particle, idx);
            double prob = get_landmark_prob(particle, idx)[0];
            if(prob <= 0) {
                remove_landmark(particle, idx);
            }
        } 
    }
}

__global__ void update(
    double *particles, int block_size, int *scratchpad_mem, int scratchpad_size, double measurements_array[][2], int n_particles, int n_measurements,
    double *measurement_cov, double threshold, double range, double fov, int max_landmarks)
{

    if(n_measurements == 0) {
        return;
    }

    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int thread_id = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int *scratchpad = scratchpad_mem + (2 * thread_id * max_landmarks);
    int *in_range = scratchpad;
    int *n_matches = in_range + max_landmarks;

    landmark_measurements measurements;
    measurements.n_measurements = n_measurements;
    measurements.measurement_cov = measurement_cov;
    measurements.measurements = measurements_array;

    for(int k = 0; k < block_size; k++) {
        int particle_id = thread_id*block_size + k;
        if(particle_id >= n_particles) {
            return;
        }
        
        double *particle = get_particle(particles, particle_id);
        int n_landmarks = get_n_landmarks(particle);
    
        if(n_landmarks == 0) {
            add_measurements_as_landmarks(particle, &measurements);
            continue;
        }

        update_landmarks(particle_id, particle, &measurements, in_range, n_matches, range, fov, threshold);
    }
}
